#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <random>

#include <thrust/merge.h>

//#define VERIFY

struct cmp_func {
    __host__ __device__
    bool operator()(int x, int y) {
        return x < y;
    }
};

int main(int argc, char **argv) {
	if (argc != 3) {
		fprintf(stderr, "Usage: %s <total number of warps (positive power of 2 required)> <RNG seed value>\n", argv[0]);
        exit(1);
	}

	int num_warps = atoi(argv[1]);
    if (num_warps <= 0) {
        fprintf(stderr, "ERROR: total number of warps must be positive!\n");
        exit(1);
    }
    //TODO: check for power of 2

    int n = num_warps * 32 * MY_E;		//w = 32
	
    //1. Generate random input
    int *input = (int *)malloc(n * sizeof(int));
    for (int i = 0; i < n; ++i) {
        input[i] = i;
    }

    std::mt19937 gen(atoi(argv[2]));
    std::uniform_int_distribution<int> dist(0, n-1);

    for (int i = 0; i < n; ++i) {
        int j = dist(gen);

        int temp = input[i];
        input[i] = input[j];
        input[j] = temp;
    }


    //2. Initialize GPU input/output arrays
    hipError_t cudaerr;
    int *d_input;
    //int *d_output;

    cudaerr = hipMalloc(&d_input,  n * sizeof(int));
    if (cudaerr != hipSuccess) printf("hipMalloc(d_input) failed with error %i \"%s\".\n", cudaerr, hipGetErrorString(cudaerr));

    //cudaerr = hipMalloc(&d_output, n * sizeof(int));
    //if (cudaerr != hipSuccess) printf("hipMalloc(d_output) failed with error %i \"%s\".\n", cudaerr, hipGetErrorString(cudaerr));

    cudaerr = hipMemcpy(d_input, input, n * sizeof(int), hipMemcpyHostToDevice);
    if (cudaerr != hipSuccess) printf("hipMemcpy(d_input, input) failed with error %i \"%s\".\n", cudaerr, hipGetErrorString(cudaerr));

    free(input);


    //3. Run merge
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    thrust::sort(thrust::device, d_input, d_input + n, cmp_func());
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("%f\n", ms);


    //4. (Optional) Verify output
    #ifdef VERIFY
    int *output = (int *)malloc(n * sizeof(int));
    cudaerr = hipMemcpy(output, d_input, n * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaerr != hipSuccess) printf("hipMemcpy(output, d_input) failed with error %i \"%s\".\n", cudaerr, hipGetErrorString(cudaerr));

    int count = 0;
    for (int i = 1; i < n; ++i) {
        if (output[i] < output[i-1]) ++count;
    }
    if (count > 0) printf("\n%d errors\n", count);
    else printf("\nsorted!\n");

    free(output);
    #endif

    //5. Clean up
    hipFree(d_input);
    //hipFree(d_output);
    
    return 0;
}

