#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <string>

#include <thrust/merge.h>

//#define VERIFY

struct cmp_func {
    __host__ __device__
    bool operator()(int x, int y) {
        return x < y;
    }
};

int main(int argc, char **argv) {
    if (argc != 3) {
		fprintf(stderr, "Usage: %s <total number of warps (positive power of 2 required)> <directory filepath>\n", argv[0]);
        exit(1);
	}

	int num_warps = atoi(argv[1]);
    if (num_warps <= 0) {
        fprintf(stderr, "ERROR: total number of warps must be positive!\n");
        exit(1);
    }
    //TODO: check for power of 2


    //1. Read asymptotic worst case input
    int n = num_warps * 32 * MY_E;        //w = 32
    int *input = (int *)malloc(n * sizeof(int));

    std::string filename = argv[2] + std::to_string(n);
    FILE *fp = fopen(filename.c_str(), "rb");
    fread(input, sizeof(int), (size_t)n, fp);
    fclose(fp);


    //2. Initialize GPU input/output arrays
    hipError_t cudaerr;
    int *d_input;
    cudaerr = hipMalloc(&d_input,  n * sizeof(int));
    if (cudaerr != hipSuccess) printf("hipMalloc(d_input) failed with error %i \"%s\".\n", cudaerr, hipGetErrorString(cudaerr));
    cudaerr = hipMemcpy(d_input, input, n * sizeof(int), hipMemcpyHostToDevice);
    if (cudaerr != hipSuccess) printf("hipMemcpy(d_input, input) failed with error %i \"%s\".\n", cudaerr, hipGetErrorString(cudaerr));
    free(input);


    //3. Run merge sort
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    thrust::sort(thrust::device, d_input, d_input + n, cmp_func());
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("%f\n", ms);


    //4. (Optional) Verify output
    #ifdef VERIFY
    int *output = (int *)malloc(n * sizeof(int));
    cudaerr = hipMemcpy(output, d_input, n * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaerr != hipSuccess) printf("hipMemcpy(output, d_input) failed with error %i \"%s\".\n", cudaerr, hipGetErrorString(cudaerr));

    int count = 0;
    for (int i = 1; i < n; ++i) {
        if (output[i] < output[i-1]) ++count;
    }
    if (count > 0) printf("\n%d errors\n", count);
    else printf("\nsorted!\n");

    free(output);
    #endif

    //5. Clean up
    hipFree(d_input);
    
    return 0;
}

